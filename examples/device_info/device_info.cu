// Copyright (c) 2020 Saurabh Yadav
// 
// This software is released under the MIT License.
// https://opensource.org/licenses/MIT

#include <stdio.h>        
#include <unistd.h> 
#include <stdlib.h> 
     
#include <hip/hip_runtime.h>

int main()
{
    hipError_t err = hipSuccess;

    int device_count;
    err = hipGetDeviceCount(&device_count);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to get device count (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Number of GPUs:\t\t %d \n",device_count);

    hipDeviceProp_t dev_prop;

    for(int i=0; i<device_count; i++) {

        err = hipGetDeviceProperties(&dev_prop, i);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to get device %d properties (error code %s)!\n", i, hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        printf("GPU Name:\t\t %s \n",dev_prop.name);
        printf("Clock rate:\t\t %d \n",dev_prop.clockRate);
        printf("Max Threads per block:\t %d \n",dev_prop.maxThreadsPerBlock);
        printf("Shared Memory per block: %lu \n",dev_prop.sharedMemPerBlock);
    }
    
    return 0;
}
